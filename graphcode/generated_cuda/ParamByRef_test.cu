// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "ParamByRef_test.h"

void test(double &arg1,int &arg2)

{

  //DECLAR DEVICE AND HOST vars in params

  //BEGIN DSL PARSING 
  int a = 0; // asst in .cu

  if (a == 0){ // if filter begin 
    arg1 = 1.000000;
    arg2 = 2;

  } // if filter end
  else
  arg1 = 2.000000;
  arg2 = 3;


  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

} //end FUN
